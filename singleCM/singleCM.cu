#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <omp.h>
#include <sys/time.h>
#include <omp.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <ctype.h>
#include "tbb/parallel_sort.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
using namespace std;
const double pi=3.141592653589793238462643383279502884197;
int ref_line[20];
char ref_file[20][16];
int sam_line[20];
char sam_file[20][16];
const int GPU_N = 2;
const int GBSize = 1024 * 1024 * 1024;
const int block_size = 512;
const int TILE_SIZE = 1024;
struct NODE
{
	double ra,dec;
	int pix;
};

bool cmp(NODE a,NODE b)
{
	return a.pix < b.pix;
}

void readFile(char *file,int N, NODE nn[])
{
	FILE *fd = fopen(file,"r");
	if(fd == NULL)
		printf("Read %s error!\n",file);
	for(int i = 0; i < N; ++i)
		fscanf(fd,"%d%lf%lf",&nn[i].pix,&nn[i].ra,&nn[i].dec);
	fclose(fd);
}
	__host__ __device__
int begin_index(int key, NODE *node, int N)
{
	for(int i = 0; i < N; ++i)
		if(node[i].pix > key)
			return i;
	return N;
}

	__host__ __device__
int binary_search(int key, NODE *node, int N)
{
	int st = 0;
	int ed = N - 1;
	while(st < ed)
	{
		int mid = st + ((ed - st) >> 1);
		if(node[mid].pix <= key)
			st = mid + 1;
		else
			ed = mid;
	}
	if(node[ed].pix > key)
		return ed;
	return -1;
}
__host__ __device__ double radians(double degree)
{
	return degree * pi / 180.0;
}
__host__	__device__
bool matched(double ra1,double dec1,double ra2,double dec2,double radius)
{
	double z1 = sin(radians(dec1));
	double x1 = cos(radians(dec1)) * cos(radians(ra1));
	double y1 = cos(radians(dec1)) * sin(radians(ra1));

	double z2 = sin(radians(dec2));
	double x2 = cos(radians(dec2)) * cos(radians(ra2));
	double y2 = cos(radians(dec2)) * sin(radians(ra2));

	double distance = (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2);
	double dist2 = 4 * pow(sin(radians(0.0056 / 2)),2);

	if(distance <= dist2)
		return true;
	return false;
}
__global__
void kernel_singleCM(NODE *ref_node, int ref_N, NODE *sam_node, int sam_N, int *sam_match,int *sam_matchedCnt,int ref_offset,int sam_offset)
{
	__shared__ int s_ref_pix[TILE_SIZE];
	__shared__ double s_ref_ra[TILE_SIZE];
	__shared__ double s_ref_dec[TILE_SIZE];

	__shared__ int start_pix,end_pix;
	__shared__ int start_ref_pos,end_ref_pos;
	__shared__ int block_sam_N,block_ref_N;
	__shared__ int iteration;

	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(threadIdx.x == 0)
	{
		if(blockIdx.x == gridDim.x - 1) // the last block
			block_sam_N = sam_N - blockIdx.x * blockDim.x;
		else
			block_sam_N = blockDim.x;

		start_pix = sam_node[tid].pix;
		end_pix = sam_node[tid + block_sam_N - 1].pix;

		if(start_pix == 0)
			start_ref_pos = 0;
		else
			start_ref_pos = binary_search(start_pix - 1,ref_node,ref_N);

		end_ref_pos = binary_search(end_pix,ref_node,ref_N);
		if(end_ref_pos == -1)
			end_ref_pos = ref_N - 1;
		else
			end_ref_pos--;
		block_ref_N = end_ref_pos - start_ref_pos + 1;
		iteration = ceil(block_ref_N * 1.0 / TILE_SIZE);
	}

	__syncthreads();
	if(start_ref_pos == -1)
		return;
	int pix,cnt = 0;
	double sam_ra,sam_dec;
	if(tid < sam_N)
	{
		pix = sam_node[tid].pix;
		sam_ra = sam_node[tid].ra;
		sam_dec = sam_node[tid].dec;
		cnt = 0;
	}

	__syncthreads();
	for(int ite = 0; ite < iteration; ++ite)
	{
		for(int k = 0; k < TILE_SIZE / blockDim.x; ++k)
		{
			int ref_pos = start_ref_pos + ite * TILE_SIZE + blockDim.x * k + threadIdx.x;
			int s_ref_pos = blockDim.x * k + threadIdx.x;
			if(ref_pos <= end_ref_pos)
			{
				s_ref_pix[s_ref_pos] = ref_node[ref_pos].pix;
				s_ref_ra[s_ref_pos] = ref_node[ref_pos].ra;
				s_ref_dec[s_ref_pos] = ref_node[ref_pos].dec;
			}
			else
				s_ref_pix[s_ref_pos] = -1;
		}

		__syncthreads();

		if(tid >= sam_N)
			continue;

		for(int j = 0; j < TILE_SIZE; ++j)
		{
			if(s_ref_pix[j] == -1 || s_ref_pix[j] > pix)
				break;
			if(s_ref_pix[j] < pix)
				continue;
			if(matched(sam_ra,sam_dec,s_ref_ra[j],s_ref_dec[j],0.0056))
			{
				cnt++;
				if(cnt <= 5)
					sam_match[tid * 5 + cnt] = ref_offset + start_ref_pos + ite * TILE_SIZE + j;
			}
		}
		__syncthreads();
	}

	sam_matchedCnt[tid] = cnt;
}


void singleCM(NODE h_ref_node[], int ref_N, NODE h_sam_node[], int sam_N, int h_sam_match[],int h_sam_matchedCnt[])
{
	//the maximum number of sample points that can be matched each time by each card
	int part_sam_N = 25000000;
	int part_ref_N = 8 * part_sam_N;

	NODE *d_ref_node[GPU_N];
	NODE *d_sam_node[GPU_N];
	int *d_sam_match[GPU_N], *d_sam_matchedCnt[GPU_N];

	omp_set_num_threads(GPU_N);
#pragma omp parallel
	{
		int i = omp_get_thread_num() % GPU_N;
		checkCudaErrors(hipSetDevice(i));
		checkCudaErrors(hipDeviceReset());

		size_t free_mem,total_mem;
		checkCudaErrors(hipMemGetInfo(&free_mem,&total_mem));
		printf("Card %d before malloc %.2lf GB, total memory %.2lf GB\n",i,free_mem * 1.0 / GBSize,total_mem * 1.0 / GBSize);


		checkCudaErrors(hipMalloc(&d_ref_node[i],sizeof(NODE) * part_ref_N));
		checkCudaErrors(hipMalloc(&d_sam_node[i],sizeof(NODE) * part_sam_N));
		checkCudaErrors(hipMalloc(&d_sam_match[i],sizeof(int) * part_sam_N  * 5));
		checkCudaErrors(hipMalloc(&d_sam_matchedCnt[i],sizeof(int) * part_sam_N));
		checkCudaErrors(hipMemset(d_sam_matchedCnt[i],0,sizeof(int) * part_sam_N));

		checkCudaErrors(hipMemGetInfo(&free_mem,&total_mem));
		printf("Card %d after malloc %.2lf GB, total memory %.2lf GB\n",i,free_mem * 1.0 / GBSize,total_mem * 1.0 / GBSize);

		//the total number of sample points processed by this card
		int card_sam_N;
		if(i == GPU_N - 1)
			card_sam_N = sam_N - i * sam_N / GPU_N;
		else
			card_sam_N = sam_N / GPU_N;

		int iteration = ceil(card_sam_N * 1.0 / part_sam_N);

		for(int ite = 0; ite < iteration; ++ite)
		{
			int cur_sam_N;
			if(ite == iteration - 1) // the last round
				cur_sam_N = card_sam_N - ite * part_sam_N;
			else
				cur_sam_N = part_sam_N;

			int start_sam_pos = ite * part_sam_N + i * sam_N / GPU_N;
			int end_sam_pos = start_sam_pos + cur_sam_N - 1;

			int start_pix = h_sam_node[start_sam_pos].pix;
			int end_pix = h_sam_node[end_sam_pos].pix;

			int start_ref_pos;
			if(start_pix == 0)
				start_ref_pos = 0;
			else
				start_ref_pos = binary_search(start_pix - 1,h_ref_node,ref_N);

			if(start_ref_pos == -1)
				break;
			int end_ref_pos = binary_search(end_pix,h_ref_node,ref_N) - 1;
			if(end_ref_pos == -2)
				end_ref_pos = ref_N - 1;
			int cur_ref_N = end_ref_pos - start_ref_pos + 1;

			dim3 block(block_size);
			dim3 grid(min(65536,(int)ceil(cur_sam_N * 1.0 / block.x)));

			printf("\n\nCard %d iteration %d\n",i,ite);
			printf("block.x %d grid.x %d\n",block.x,grid.x);
			printf("start_sam_pos %d start_sam_pix %d end_sam_pos %d end_sam_pix %d sam_N %d\n",start_sam_pos,start_pix,end_sam_pos,end_pix,cur_sam_N);
			printf("start_ref_pos %d start_ref_pix %d end_ref_pos %d end_ref_pix %d ref_N %d\n",start_ref_pos,h_ref_node[start_ref_pos].pix,end_ref_pos,h_ref_node[end_ref_pos].pix,cur_ref_N);

			checkCudaErrors(hipMemcpy(d_sam_node[i],h_sam_node + start_sam_pos,cur_sam_N * sizeof(NODE),hipMemcpyHostToDevice));
			checkCudaErrors(hipMemcpy(d_ref_node[i],h_ref_node + start_ref_pos,cur_ref_N * sizeof(NODE), hipMemcpyHostToDevice));
			kernel_singleCM<<<grid,block>>>(d_ref_node[i],cur_ref_N,d_sam_node[i],cur_sam_N,d_sam_match[i],d_sam_matchedCnt[i],start_ref_pos,start_sam_pos);
			checkCudaErrors(hipMemcpy(h_sam_matchedCnt + start_sam_pos,d_sam_matchedCnt[i],cur_sam_N * sizeof(int),hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(h_sam_match + start_sam_pos * 5,d_sam_match[i],cur_sam_N * 5 * sizeof(int),hipMemcpyDeviceToHost));
		}
	}
}


int main(int argc, char *argv[])
{

	const int ref_N = 1538557732;
	const int sam_N = 200003876;
	time_t rawtime;

	FILE *fd = fopen(argv[1],"r");
	for(int i = 0; i < 20; ++i)
		fscanf(fd,"%d%s",&ref_line[i],ref_file[i]);
	fclose(fd);

	fd = fopen(argv[2],"r");
	for(int i = 0; i < 20; ++i)
		fscanf(fd,"%d%s",&sam_line[i],sam_file[i]);
	fclose(fd);

	NODE *ref_node,*sam_node;
	int *sam_matchedCnt;
	int *sam_match;

	ref_node = (NODE *)malloc(sizeof(NODE) * ref_N);
	sam_node = (NODE *)malloc(sizeof(NODE) * sam_N);

	sam_matchedCnt = (int *)malloc(sizeof(int) * sam_N);
	sam_match = (int *)malloc(sizeof(int) * sam_N * 5);


	time(&rawtime);
	printf("before read ref file : %s\n",ctime(&rawtime));

	omp_set_num_threads(20);
#pragma omp parallel
	{
		int i = omp_get_thread_num() % 20;
		int offset = i * ref_line[0];
		readFile(ref_file[i],ref_line[i],ref_node + offset);
	}

	time(&rawtime);
	printf("after read ref file : %s\n",ctime(&rawtime));

#pragma omp parallel
	{
		int i = omp_get_thread_num() % 20;
		int offset = i * sam_line[0];
		readFile(sam_file[i],sam_line[i],sam_node + offset);
	}

	time(&rawtime);
	printf("after read sam file : %s\n",ctime(&rawtime));

	tbb::parallel_sort(ref_node,ref_node + ref_N,cmp);

	time(&rawtime);
	printf("after sort : %s\n",ctime(&rawtime));


	/*
	int size = 600;
	int cnt[600];
	memset(cnt,0,sizeof(cnt));
	int cnt_j = 0;
	for(int i = 0; i < ref_N; ++i)
	{
		if(ref_node[i].pix > 0)
		{
			cout << "cur i " << i << endl;
			break;
		}
		for(int j = 0; j < sam_N; ++j)
		{
			if(sam_node[j].pix > 0)
			{
				cnt_j = j;
				break;
			}
			if(matched(sam_node[j].ra,sam_node[j].dec,ref_node[i].ra,ref_node[i].dec,0.0056))
			{
			//	printf("sam %d ra %.6lf dec %.6lf ref %d ra %.6lf dec %.6lf\n",j,sam_node[j].ra,sam_node[j].dec,i,ref_node[i].ra,ref_node[i].dec);
				cnt[i]++;
			}
		}
	}
	cout << "cnt_j " << cnt_j << endl;
	for(int i = 0; i < 515; ++i)
		cout << i << " " << cnt[i] << endl;
	return 0;
*/


	singleCM(ref_node,ref_N,sam_node,sam_N,sam_match,sam_matchedCnt);
	time(&rawtime);
	printf("singleCM : %s\n",ctime(&rawtime));
}
