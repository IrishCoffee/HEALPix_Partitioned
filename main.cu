#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <thrust/sort.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#include <omp.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include "mpi.h"
#include "tbb/parallel_sort.h"
#include "printResult.h"
#include "kernel_functions.h"
#include "helper_functions.h"
#include "values.h"
#include "worker.h"
#include "master.h"
using namespace std;

int main(int argc, char* argv[])
{
	struct timeval start,end;
	MPI_Init(&argc,&argv);
	MPI_Comm_size(MPI_COMM_WORLD,&numprocs);
	MPI_Comm_rank(MPI_COMM_WORLD,&rank);
	MPI_Get_processor_name(processor_name,&namelen);

	printf("--------------\nRank %d Processor_name %s\n------------------\n",rank,processor_name);

	if(rank == MASTER_NODE)
	{
		return 0;
		master_allocation();

		gettimeofday(&start,NULL);
		master_load_file(argv[2]);
		gettimeofday(&end,NULL);
		printf("master_load_file %.3f s \n", diffTime(start,end) * 0.001 );

		gettimeofday(&start,NULL);
		master_getPix();
		gettimeofday(&end,NULL);
		printf("master_getPix %.3f s \n", diffTime(start,end) * 0.001 );

		gettimeofday(&start,NULL);
		tbb::parallel_sort(h_sam_node,h_sam_node + sam_N,cmp);
		gettimeofday(&end,NULL);
		printf("master sort %.3f s \n", diffTime(start,end) * 0.001 );

		master_free();
	}
	else
	{
		printf("Number of host CPUs:\t%d\n",omp_get_num_procs());
		//	checkCudaErrors(hipGetDeviceCount(&GPU_N));
		printf("\n=============================\nCUDA-capable device count: %d\n",GPU_N);

		for(int i = 0; i < GPU_N; ++i)
		{
			checkCudaErrors(hipGetDeviceProperties(&deviceProp,i));
			printf("Device %d: \"%s\"\n",i,deviceProp.name);
			checkCudaErrors(hipSetDevice(i));
			checkCudaErrors(hipDeviceReset());
		}
		printf("===========================\n");
		readRefFile(argv[1],12);
		//0.2B objects
		ref_N = 200000000;

		gettimeofday(&start,NULL);
		worker_memory_allocation();
		gettimeofday(&end,NULL);
		printf("worker_memory_allocation %.3f s \n", diffTime(start,end) * 0.001 );

		gettimeofday(&start,NULL);
		worker_load_file(0);
		gettimeofday(&end,NULL);
		printf("worker_load_file %.3f s \n", diffTime(start,end) * 0.001 );

		gettimeofday(&start,NULL);
		worker_computeSI(search_radius);
		gettimeofday(&end,NULL);
		printf("worker_computeSI %.3f s \n", diffTime(start,end) * 0.001 );

		ref_dup_N = 0;
		gettimeofday(&start,NULL);
		worker_duplicateR();
		gettimeofday(&end,NULL);
		printf("worker_duplicateR %.3f s \n", diffTime(start,end) * 0.001 );

		gettimeofday(&start,NULL);
		tbb::parallel_sort(h_ref_dup_node,h_ref_dup_node + ref_dup_N,cmp);
		gettimeofday(&end,NULL);
		printf("tbb sort R by key %.3f s \n", diffTime(start,end) * 0.001 );

		gettimeofday(&start,NULL);
		worker_countR();
		gettimeofday(&end,NULL);
		printf("worker_countR %.3f s \n", diffTime(start,end) * 0.001 );

		int zeroCnt = 0;
		for(int i = 0; i < cntSize; ++i)
			if(h_R_cnt[i] == 0)
				zeroCnt++;
		printf("zeroCnt %d\n",zeroCnt);

		for(int i = 0; i < 200; ++i)
			printf("%d %.3lf %.3lf %d\n",i,h_ref_dup_node[i].ra,h_ref_dup_node[i].dec,h_ref_dup_node[i].pix);

		for(int i = 0; i < 20; ++i)
			printf("pix %d cnt %d startPos %d\n",i,h_R_cnt[i],h_R_startPos[i]);


		worker_memory_free();
	}
}
