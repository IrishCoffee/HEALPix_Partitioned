#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <thrust/sort.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#include <omp.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include "mpi.h"
#include "tbb/parallel_sort.h"
#include "printResult.h"
#include "kernel_functions.h"
#include "helper_functions.h"
#include "values.h"
#include "worker.h"
#include <fstream>
using namespace std;

int main(int argc, char* argv[])
{
	struct timeval start,end;
	//MPI Communication Initalization
	MPI_Init(&argc,&argv);
	MPI_Comm_size(MPI_COMM_WORLD,&numprocs);
	MPI_Comm_rank(MPI_COMM_WORLD,&rank);
	MPI_Get_processor_name(processor_name,&namelen);

	//Register new data type
	MPI_Datatype old_types[3];
	MPI_Aint indices[3];
	int blocklens[3];
	blocklens[0] = 1;
	blocklens[1] = 1;
	blocklens[2] = 1;
	old_types[0] = MPI_DOUBLE;
	old_types[1] = MPI_DOUBLE;
	old_types[2] = MPI_INT;

	indices[0] = 0;
	indices[1] = sizeof(double);
	indices[2] = 2 * sizeof(double);

	MPI_Type_struct(3,blocklens,indices,old_types,&mpi_node);
	MPI_Type_commit(&mpi_node);


//	freopen(processor_name,"w",stdout);
	char *referenceTable = argv[1];
	int ref_file_num = atoi(argv[2]);
	int ref_file_size = atoi(argv[3]);
	int ref_file_ignore = atoi(argv[4]);
	char *sampleTable = argv[5];
	int sam_file_num = atoi(argv[6]);
	int sam_file_size = atoi(argv[7]);
	int sam_file_ignore = atoi(argv[8]);

	ref_N = ref_file_num * ref_file_size / numprocs;
	sam_N = sam_file_num * sam_file_size / numprocs;

	cout << "ref_N " << sam_N << endl;

	time_t rawtime;
	time(&rawtime);
	printf("--------------\nRank %d Processor_name %s\n------------------\n",rank,processor_name);
	printf("%s starts at %s\n",processor_name,ctime(&rawtime));

	mem_allo(ref_file_num * ref_file_size / numprocs, sam_file_num * sam_file_size / numprocs);
	load_file_list(referenceTable,ref_file_num,sampleTable,sam_file_num);

	gettimeofday(&start,NULL);
	load_ref_file(rank,ref_file_num,ref_file_size,ref_file_ignore);
	gettimeofday(&end,NULL);
	printf("rank-%d load_ref_file %.3f s\n",rank,diffTime(start,end) * 0.001);

	gettimeofday(&start,NULL);
	load_sam_file(rank,sam_file_num,sam_file_size,sam_file_ignore);
	gettimeofday(&end,NULL);
	printf("rank-%d load_sam_file %.3f s\n",rank,diffTime(start,end) * 0.001);

	gettimeofday(&start,NULL);
	computeSI(search_radius);
	gettimeofday(&end,NULL);
	printf("rank-%d computeSI %.3f s\n",rank,diffTime(start,end) * 0.001);

	gettimeofday(&start,NULL);
	indexSample();
	gettimeofday(&end,NULL);
	printf("rank-%d indexSample %.3f s\n",rank,diffTime(start,end) * 0.001);

	gettimeofday(&start,NULL);
	tbb::parallel_sort(h_sam_node,h_sam_node + sam_N,cmp);
	gettimeofday(&end,NULL);
	printf("rank-%d sort sample %.3f s\n",rank,diffTime(start,end) * 0.001);

	gettimeofday(&start,NULL);
	count_ref(rank);
	gettimeofday(&end,NULL);
	printf("rank-%d count_ref %.3f s\n",rank,diffTime(start,end) * 0.001);

	gettimeofday(&start,NULL);
	worker_gather(rank);
	gettimeofday(&end,NULL);
	printf("rank-%d worker_gather %.3f s\n",rank,diffTime(start,end) * 0.001);

	gettimeofday(&start,NULL);
	redistribute_R(rank);
	gettimeofday(&end,NULL);
	printf("rank-%d redistribute_R %.3f s\n",rank,diffTime(start,end) * 0.001);

	time(&rawtime);
	printf("%s ends at %s\n",processor_name,ctime(&rawtime));

}
