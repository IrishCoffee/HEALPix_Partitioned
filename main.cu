#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <thrust/sort.h>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#include <omp.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include "tbb/parallel_sort.h"
#include "printResult.h"
#include "kernel_functions.h"
#include "helper_functions.h"
#include "values.h"
#include "worker.h"
using namespace std;

int main(int argc, char* argv[])
{
	struct timeval start,end;

	double search_radius = 0.0056 * pi / 180.0;
	hipDeviceProp_t deviceProp;

	printf("Number of host CPUs:\t%d\n",omp_get_num_procs());
//	checkCudaErrors(hipGetDeviceCount(&GPU_N));
	printf("\n=============================\nCUDA-capable device count: %d\n",GPU_N);

	for(int i = 0; i < GPU_N; ++i)
	{
		checkCudaErrors(hipGetDeviceProperties(&deviceProp,i));
		printf("Device %d: \"%s\"\n",i,deviceProp.name);
		checkCudaErrors(hipSetDevice(i));
		checkCudaErrors(hipDeviceReset());
	}
	printf("===========================\n");
	
	readRefFile(argv[1],12);
	//0.2B objects
	ref_N = 200000000;
	
	gettimeofday(&start,NULL);
	worker_memory_allocation();
	gettimeofday(&end,NULL);
	printf("worker_memory_allocation %.3f s \n", diffTime(start,end) * 0.001 );

	gettimeofday(&start,NULL);
	worker_load_file(0);
	gettimeofday(&end,NULL);
	printf("worker_load_file %.3f s \n", diffTime(start,end) * 0.001 );
	
	gettimeofday(&start,NULL);
	worker_computeSI(search_radius);
	gettimeofday(&end,NULL);
	printf("worker_computeSI %.3f s \n", diffTime(start,end) * 0.001 );
	
	ref_dup_N = 0;
	gettimeofday(&start,NULL);
	worker_duplicateR();
	gettimeofday(&end,NULL);
	printf("worker_duplicateR %.3f s \n", diffTime(start,end) * 0.001 );

	gettimeofday(&start,NULL);
	tbb::parallel_sort(ref_dup_node,ref_dup_node + ref_dup_N,cmp);
	gettimeofday(&end,NULL);
	printf("worker_sort %.3f s \n", diffTime(start,end) * 0.001 );
	
	worker_memory_free();

}
